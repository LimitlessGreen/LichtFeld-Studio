#include "hip/hip_runtime.h"
/* SPDX-FileCopyrightText: 2025 LichtFeld Studio Authors
 *
 * SPDX-License-Identifier: GPL-3.0-or-later */

#include "forward.h"
#include "rasterization_api_tensor.h"
#include "rasterization_config.h"
#include <functional>
#include <stdexcept>
#include <tuple>

namespace lfs::rendering {

    // Helper to create resize function for custom Tensor
    inline std::function<char*(size_t)> resize_function_wrapper_tensor(Tensor& t) {
        return [&t](size_t N) -> char* {
            if (N == 0) {
                t = Tensor::empty({0}, lfs::core::Device::CUDA, lfs::core::DataType::UInt8);
                return nullptr;
            }
            t = Tensor::empty({N}, lfs::core::Device::CUDA, lfs::core::DataType::UInt8);
            return reinterpret_cast<char*>(t.ptr<uint8_t>());
        };
    }

    // Validation for custom Tensor (similar to CHECK_INPUT)
    inline void check_tensor_input(bool debug, const Tensor& tensor, const char* name) {
        if (debug) {
            if (!tensor.is_valid() || tensor.device() != lfs::core::Device::CUDA ||
                tensor.dtype() != lfs::core::DataType::Float32 || !tensor.is_contiguous()) {
                throw std::runtime_error("Input tensor '" + std::string(name) +
                                         "' must be a contiguous CUDA float tensor.");
            }
        }
    }

    std::tuple<Tensor, Tensor>
    forward_wrapper_tensor(
        const Tensor& means,
        const Tensor& scales_raw,
        const Tensor& rotations_raw,
        const Tensor& opacities_raw,
        const Tensor& sh_coefficients_0,
        const Tensor& sh_coefficients_rest,
        const Tensor& w2c,
        const Tensor& cam_position,
        const int active_sh_bases,
        const int width,
        const int height,
        const float focal_x,
        const float focal_y,
        const float center_x,
        const float center_y,
        const float near_plane,
        const float far_plane) {

        // Validate all input tensors
        check_tensor_input(config::debug, means, "means");
        check_tensor_input(config::debug, scales_raw, "scales_raw");
        check_tensor_input(config::debug, rotations_raw, "rotations_raw");
        check_tensor_input(config::debug, opacities_raw, "opacities_raw");
        check_tensor_input(config::debug, sh_coefficients_0, "sh_coefficients_0");
        check_tensor_input(config::debug, sh_coefficients_rest, "sh_coefficients_rest");

        // Extract dimensions
        const int n_primitives = static_cast<int>(means.size(0));
        const int total_bases_sh_rest = static_cast<int>(sh_coefficients_rest.size(1));

        // Allocate output tensors
        Tensor image = Tensor::empty({3, static_cast<size_t>(height), static_cast<size_t>(width)},
                                     lfs::core::Device::CUDA, lfs::core::DataType::Float32);
        Tensor alpha = Tensor::empty({1, static_cast<size_t>(height), static_cast<size_t>(width)},
                                     lfs::core::Device::CUDA, lfs::core::DataType::Float32);

        // Create buffer tensors (these will be resized by the forward function)
        Tensor per_primitive_buffers = Tensor::empty({0}, lfs::core::Device::CUDA, lfs::core::DataType::UInt8);
        Tensor per_tile_buffers = Tensor::empty({0}, lfs::core::Device::CUDA, lfs::core::DataType::UInt8);
        Tensor per_instance_buffers = Tensor::empty({0}, lfs::core::Device::CUDA, lfs::core::DataType::UInt8);

        // Create allocator functions
        const std::function<char*(size_t)> per_primitive_buffers_func =
            resize_function_wrapper_tensor(per_primitive_buffers);
        const std::function<char*(size_t)> per_tile_buffers_func =
            resize_function_wrapper_tensor(per_tile_buffers);
        const std::function<char*(size_t)> per_instance_buffers_func =
            resize_function_wrapper_tensor(per_instance_buffers);

        // Ensure w2c and cam_position are contiguous
        Tensor w2c_contig = w2c.is_contiguous() ? w2c : w2c.contiguous();
        Tensor cam_pos_contig = cam_position.is_contiguous() ? cam_position : cam_position.contiguous();

        // Call the actual CUDA forward function
        forward(
            per_primitive_buffers_func,
            per_tile_buffers_func,
            per_instance_buffers_func,
            reinterpret_cast<const float3*>(means.ptr<float>()),
            reinterpret_cast<const float3*>(scales_raw.ptr<float>()),
            reinterpret_cast<const float4*>(rotations_raw.ptr<float>()),
            opacities_raw.ptr<float>(),
            reinterpret_cast<const float3*>(sh_coefficients_0.ptr<float>()),
            reinterpret_cast<const float3*>(sh_coefficients_rest.ptr<float>()),
            reinterpret_cast<const float4*>(w2c_contig.ptr<float>()),
            reinterpret_cast<const float3*>(cam_pos_contig.ptr<float>()),
            image.ptr<float>(),
            alpha.ptr<float>(),
            n_primitives,
            active_sh_bases,
            total_bases_sh_rest,
            width,
            height,
            focal_x,
            focal_y,
            center_x,
            center_y,
            near_plane,
            far_plane);

        return {std::move(image), std::move(alpha)};
    }

} // namespace lfs::rendering
