#include "hip/hip_runtime.h"
/* SPDX-FileCopyrightText: 2025 LichtFeld Studio Authors
 * SPDX-License-Identifier: GPL-3.0-or-later */

#include "core_new/logger.hpp"
#include "internal/memory_pool.hpp"
#include "internal/tensor_functors.hpp"
#include "internal/tensor_ops.hpp"
#include <hip/hip_runtime.h>
#include <thrust/copy.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/transform_iterator.h>

namespace lfs::core::tensor_ops {

    // Note: run_with_thrust_policy is now in include/core/tensor_generic_ops.cuh

    // ============================================================================
    // BROADCASTING INDEX FUNCTOR (for single-array broadcast)
    // ============================================================================

    template <int MaxRank = 8>
    struct broadcast_index_functor {
        int src_rank, dst_rank;
        int src_shape[MaxRank];
        int dst_shape[MaxRank];
        int src_strides[MaxRank];
        int dst_strides[MaxRank];

        broadcast_index_functor(const std::vector<size_t>& src_shape_vec,
                                const std::vector<size_t>& dst_shape_vec)
            : src_rank(src_shape_vec.size()),
              dst_rank(dst_shape_vec.size()) {

            for (int i = 0; i < src_rank; ++i) {
                src_shape[i] = static_cast<int>(src_shape_vec[i]);
            }
            for (int i = 0; i < dst_rank; ++i) {
                dst_shape[i] = static_cast<int>(dst_shape_vec[i]);
            }

            // Compute row-major strides
            if (src_rank > 0) {
                src_strides[src_rank - 1] = 1;
                for (int i = src_rank - 2; i >= 0; --i) {
                    src_strides[i] = src_strides[i + 1] * src_shape[i + 1];
                }
            }

            if (dst_rank > 0) {
                dst_strides[dst_rank - 1] = 1;
                for (int i = dst_rank - 2; i >= 0; --i) {
                    dst_strides[i] = dst_strides[i + 1] * dst_shape[i + 1];
                }
            }
        }

        __device__ size_t operator()(size_t dst_linear_idx) const {
            size_t src_idx = 0;
            size_t remaining = dst_linear_idx;

            for (int i = 0; i < dst_rank; ++i) {
                int dst_coord = remaining / dst_strides[i];
                remaining %= dst_strides[i];

                int offset = dst_rank - src_rank;
                if (i >= offset) {
                    int src_dim = i - offset;
                    int src_coord = (src_shape[src_dim] == 1) ? 0 : dst_coord;
                    src_idx += src_coord * src_strides[src_dim];
                }
            }

            return src_idx;
        }
    };

    // ============================================================================
    // SINGLE-ARRAY BROADCASTING (Generic) - NOT used by binary ops
    // ============================================================================

    template <typename T>
    void launch_broadcast_generic(const T* src, T* dst,
                                  const size_t* src_shape, const size_t* dst_shape,
                                  size_t src_rank, size_t dst_rank,
                                  size_t dst_elements, hipStream_t stream) {
        if (dst_elements == 0)
            return;

        std::vector<size_t> src_vec(src_shape, src_shape + src_rank);
        std::vector<size_t> dst_vec(dst_shape, dst_shape + dst_rank);

        auto src_ptr = thrust::device_pointer_cast(src);
        auto dst_ptr = thrust::device_pointer_cast(dst);

        broadcast_index_functor<> index_mapper(src_vec, dst_vec);

        auto counting = thrust::make_counting_iterator<size_t>(0);
        auto src_index_iter = thrust::make_transform_iterator(counting, index_mapper);
        auto permuted_src = thrust::make_permutation_iterator(src_ptr, src_index_iter);

        run_with_thrust_policy(stream, [&](auto policy) {
            thrust::copy(policy, permuted_src, permuted_src + dst_elements, dst_ptr);
        });
    }

    void launch_broadcast(const float* src, float* dst,
                          const size_t* src_shape, const size_t* dst_shape,
                          size_t src_rank, size_t dst_rank,
                          size_t dst_elements, hipStream_t stream) {
        launch_broadcast_generic(src, dst, src_shape, dst_shape, src_rank, dst_rank, dst_elements, stream);
    }

    void launch_broadcast_bool(const unsigned char* src, unsigned char* dst,
                               const size_t* src_shape, const size_t* dst_shape,
                               size_t src_rank, size_t dst_rank,
                               size_t dst_elements, hipStream_t stream) {
        launch_broadcast_generic(src, dst, src_shape, dst_shape, src_rank, dst_rank, dst_elements, stream);
    }

    // ============================================================================
    // NOTE: launch_broadcast_binary implementation is now in tensor_broadcast_ops.cuh
    // All CUDA kernels and the host function template are defined inline in the header
    // for correct template instantiation with expression template functors.
    // ============================================================================

    // ============================================================================
    // EXPLICIT INSTANTIATIONS FOR C++ FILES
    // C++ files can't see tensor_broadcast_ops.cuh (which is #ifdef __HIPCC__),
    // so we need explicit instantiations for basic binary operations.
    // ============================================================================

    // Arithmetic operations (same input/output type - comprehensive list)
    template void launch_broadcast_binary<float, float, ops::add_op>(
        const float*, const float*, float*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::add_op, hipStream_t);

    template void launch_broadcast_binary<int, int, ops::add_op>(
        const int*, const int*, int*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::add_op, hipStream_t);

    template void launch_broadcast_binary<float, float, ops::sub_op>(
        const float*, const float*, float*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::sub_op, hipStream_t);

    template void launch_broadcast_binary<int, int, ops::sub_op>(
        const int*, const int*, int*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::sub_op, hipStream_t);

    template void launch_broadcast_binary<float, float, ops::mul_op>(
        const float*, const float*, float*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::mul_op, hipStream_t);

    template void launch_broadcast_binary<int, int, ops::mul_op>(
        const int*, const int*, int*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::mul_op, hipStream_t);

    template void launch_broadcast_binary<float, float, ops::div_op>(
        const float*, const float*, float*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::div_op, hipStream_t);

    template void launch_broadcast_binary<int, int, ops::div_op>(
        const int*, const int*, int*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::div_op, hipStream_t);

    // Comparison operations (input T -> output unsigned char/bool)
    template void launch_broadcast_binary<float, unsigned char, ops::greater_op>(
        const float*, const float*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::greater_op, hipStream_t);

    template void launch_broadcast_binary<int, unsigned char, ops::greater_op>(
        const int*, const int*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::greater_op, hipStream_t);

    template void launch_broadcast_binary<unsigned char, unsigned char, ops::greater_op>(
        const unsigned char*, const unsigned char*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::greater_op, hipStream_t);

    template void launch_broadcast_binary<float, unsigned char, ops::greater_equal_op>(
        const float*, const float*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::greater_equal_op, hipStream_t);

    template void launch_broadcast_binary<int, unsigned char, ops::greater_equal_op>(
        const int*, const int*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::greater_equal_op, hipStream_t);

    template void launch_broadcast_binary<unsigned char, unsigned char, ops::greater_equal_op>(
        const unsigned char*, const unsigned char*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::greater_equal_op, hipStream_t);

    template void launch_broadcast_binary<float, unsigned char, ops::less_equal_op>(
        const float*, const float*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::less_equal_op, hipStream_t);

    template void launch_broadcast_binary<int, unsigned char, ops::less_equal_op>(
        const int*, const int*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::less_equal_op, hipStream_t);

    template void launch_broadcast_binary<unsigned char, unsigned char, ops::less_equal_op>(
        const unsigned char*, const unsigned char*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::less_equal_op, hipStream_t);

    template void launch_broadcast_binary<float, unsigned char, ops::less_op>(
        const float*, const float*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::less_op, hipStream_t);

    template void launch_broadcast_binary<int, unsigned char, ops::less_op>(
        const int*, const int*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::less_op, hipStream_t);

    template void launch_broadcast_binary<unsigned char, unsigned char, ops::less_op>(
        const unsigned char*, const unsigned char*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::less_op, hipStream_t);

    template void launch_broadcast_binary<float, unsigned char, ops::equal_op>(
        const float*, const float*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::equal_op, hipStream_t);

    template void launch_broadcast_binary<int, unsigned char, ops::equal_op>(
        const int*, const int*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::equal_op, hipStream_t);

    template void launch_broadcast_binary<unsigned char, unsigned char, ops::equal_op>(
        const unsigned char*, const unsigned char*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::equal_op, hipStream_t);

    // Logical operations (bool/unsigned char -> unsigned char)
    template void launch_broadcast_binary<float, unsigned char, ops::logical_and_op>(
        const float*, const float*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::logical_and_op, hipStream_t);

    template void launch_broadcast_binary<int, unsigned char, ops::logical_and_op>(
        const int*, const int*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::logical_and_op, hipStream_t);

    template void launch_broadcast_binary<unsigned char, unsigned char, ops::logical_and_op>(
        const unsigned char*, const unsigned char*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::logical_and_op, hipStream_t);

    template void launch_broadcast_binary<float, unsigned char, ops::logical_or_op>(
        const float*, const float*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::logical_or_op, hipStream_t);

    template void launch_broadcast_binary<int, unsigned char, ops::logical_or_op>(
        const int*, const int*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::logical_or_op, hipStream_t);

    template void launch_broadcast_binary<unsigned char, unsigned char, ops::logical_or_op>(
        const unsigned char*, const unsigned char*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::logical_or_op, hipStream_t);

    // Min/max operations
    template void launch_broadcast_binary<float, float, ops::minimum_op>(
        const float*, const float*, float*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::minimum_op, hipStream_t);

    template void launch_broadcast_binary<int, int, ops::minimum_op>(
        const int*, const int*, int*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::minimum_op, hipStream_t);

    template void launch_broadcast_binary<float, float, ops::maximum_op>(
        const float*, const float*, float*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::maximum_op, hipStream_t);

    template void launch_broadcast_binary<int, int, ops::maximum_op>(
        const int*, const int*, int*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::maximum_op, hipStream_t);

    // Power operations
    template void launch_broadcast_binary<float, float, ops::pow_op>(
        const float*, const float*, float*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::pow_op, hipStream_t);

    template void launch_broadcast_binary<int, int, ops::pow_op>(
        const int*, const int*, int*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::pow_op, hipStream_t);

    // Not equal operation
    template void launch_broadcast_binary<float, unsigned char, ops::not_equal_op>(
        const float*, const float*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::not_equal_op, hipStream_t);

    template void launch_broadcast_binary<int, unsigned char, ops::not_equal_op>(
        const int*, const int*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::not_equal_op, hipStream_t);

    template void launch_broadcast_binary<unsigned char, unsigned char, ops::not_equal_op>(
        const unsigned char*, const unsigned char*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::not_equal_op, hipStream_t);

    // ============================================================================
    // Type Promotion Broadcast Instantiations
    // ============================================================================
    // Added to support the type promotion system for mixed-dtype operations
    // with broadcasting.
    // ============================================================================

    // Float16 broadcast operations
    template void launch_broadcast_binary<__half, __half, ops::add_op>(
        const __half*, const __half*, __half*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::add_op, hipStream_t);
    template void launch_broadcast_binary<__half, __half, ops::sub_op>(
        const __half*, const __half*, __half*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::sub_op, hipStream_t);
    template void launch_broadcast_binary<__half, __half, ops::mul_op>(
        const __half*, const __half*, __half*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::mul_op, hipStream_t);
    template void launch_broadcast_binary<__half, __half, ops::div_op>(
        const __half*, const __half*, __half*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::div_op, hipStream_t);
    template void launch_broadcast_binary<__half, __half, ops::maximum_op>(
        const __half*, const __half*, __half*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::maximum_op, hipStream_t);
    template void launch_broadcast_binary<__half, __half, ops::minimum_op>(
        const __half*, const __half*, __half*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::minimum_op, hipStream_t);
    template void launch_broadcast_binary<__half, __half, ops::pow_op>(
        const __half*, const __half*, __half*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::pow_op, hipStream_t);

    // Int64 broadcast operations
    template void launch_broadcast_binary<int64_t, int64_t, ops::add_op>(
        const int64_t*, const int64_t*, int64_t*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::add_op, hipStream_t);
    template void launch_broadcast_binary<int64_t, int64_t, ops::sub_op>(
        const int64_t*, const int64_t*, int64_t*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::sub_op, hipStream_t);
    template void launch_broadcast_binary<int64_t, int64_t, ops::mul_op>(
        const int64_t*, const int64_t*, int64_t*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::mul_op, hipStream_t);
    template void launch_broadcast_binary<int64_t, int64_t, ops::div_op>(
        const int64_t*, const int64_t*, int64_t*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::div_op, hipStream_t);
    template void launch_broadcast_binary<int64_t, int64_t, ops::maximum_op>(
        const int64_t*, const int64_t*, int64_t*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::maximum_op, hipStream_t);
    template void launch_broadcast_binary<int64_t, int64_t, ops::minimum_op>(
        const int64_t*, const int64_t*, int64_t*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::minimum_op, hipStream_t);
    template void launch_broadcast_binary<int64_t, int64_t, ops::pow_op>(
        const int64_t*, const int64_t*, int64_t*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::pow_op, hipStream_t);
    template void launch_broadcast_binary<int64_t, int64_t, ops::mod_op>(
        const int64_t*, const int64_t*, int64_t*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::mod_op, hipStream_t);

    // UInt8 broadcast operations
    template void launch_broadcast_binary<uint8_t, uint8_t, ops::add_op>(
        const uint8_t*, const uint8_t*, uint8_t*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::add_op, hipStream_t);
    template void launch_broadcast_binary<uint8_t, uint8_t, ops::sub_op>(
        const uint8_t*, const uint8_t*, uint8_t*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::sub_op, hipStream_t);
    template void launch_broadcast_binary<uint8_t, uint8_t, ops::mul_op>(
        const uint8_t*, const uint8_t*, uint8_t*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::mul_op, hipStream_t);
    template void launch_broadcast_binary<uint8_t, uint8_t, ops::div_op>(
        const uint8_t*, const uint8_t*, uint8_t*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::div_op, hipStream_t);
    template void launch_broadcast_binary<uint8_t, uint8_t, ops::maximum_op>(
        const uint8_t*, const uint8_t*, uint8_t*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::maximum_op, hipStream_t);
    template void launch_broadcast_binary<uint8_t, uint8_t, ops::minimum_op>(
        const uint8_t*, const uint8_t*, uint8_t*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::minimum_op, hipStream_t);
    template void launch_broadcast_binary<uint8_t, uint8_t, ops::pow_op>(
        const uint8_t*, const uint8_t*, uint8_t*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::pow_op, hipStream_t);

    // mod_op broadcast (was missing!)
    template void launch_broadcast_binary<float, float, ops::mod_op>(
        const float*, const float*, float*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::mod_op, hipStream_t);
    template void launch_broadcast_binary<int, int, ops::mod_op>(
        const int*, const int*, int*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::mod_op, hipStream_t);

    // Comparison operations for additional types
    template void launch_broadcast_binary<int64_t, unsigned char, ops::greater_op>(
        const int64_t*, const int64_t*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::greater_op, hipStream_t);
    template void launch_broadcast_binary<int64_t, unsigned char, ops::greater_equal_op>(
        const int64_t*, const int64_t*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::greater_equal_op, hipStream_t);
    template void launch_broadcast_binary<int64_t, unsigned char, ops::less_op>(
        const int64_t*, const int64_t*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::less_op, hipStream_t);
    template void launch_broadcast_binary<int64_t, unsigned char, ops::less_equal_op>(
        const int64_t*, const int64_t*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::less_equal_op, hipStream_t);
    template void launch_broadcast_binary<int64_t, unsigned char, ops::equal_op>(
        const int64_t*, const int64_t*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::equal_op, hipStream_t);
    template void launch_broadcast_binary<int64_t, unsigned char, ops::not_equal_op>(
        const int64_t*, const int64_t*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::not_equal_op, hipStream_t);

    template void launch_broadcast_binary<__half, unsigned char, ops::greater_op>(
        const __half*, const __half*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::greater_op, hipStream_t);
    template void launch_broadcast_binary<__half, unsigned char, ops::greater_equal_op>(
        const __half*, const __half*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::greater_equal_op, hipStream_t);
    template void launch_broadcast_binary<__half, unsigned char, ops::less_op>(
        const __half*, const __half*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::less_op, hipStream_t);
    template void launch_broadcast_binary<__half, unsigned char, ops::less_equal_op>(
        const __half*, const __half*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::less_equal_op, hipStream_t);
    template void launch_broadcast_binary<__half, unsigned char, ops::equal_op>(
        const __half*, const __half*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::equal_op, hipStream_t);
    template void launch_broadcast_binary<__half, unsigned char, ops::not_equal_op>(
        const __half*, const __half*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::not_equal_op, hipStream_t);

    // Logical operations for additional types
    template void launch_broadcast_binary<int64_t, unsigned char, ops::logical_and_op>(
        const int64_t*, const int64_t*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::logical_and_op, hipStream_t);
    template void launch_broadcast_binary<int64_t, unsigned char, ops::logical_or_op>(
        const int64_t*, const int64_t*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::logical_or_op, hipStream_t);
    template void launch_broadcast_binary<__half, unsigned char, ops::logical_and_op>(
        const __half*, const __half*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::logical_and_op, hipStream_t);
    template void launch_broadcast_binary<__half, unsigned char, ops::logical_or_op>(
        const __half*, const __half*, unsigned char*,
        const size_t*, const size_t*, const size_t*,
        size_t, size_t, size_t, size_t, ops::logical_or_op, hipStream_t);

} // namespace lfs::core::tensor_ops
