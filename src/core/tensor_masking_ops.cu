#include "hip/hip_runtime.h"
/* SPDX-FileCopyrightText: 2025 LichtFeld Studio Authors
 * SPDX-License-Identifier: GPL-3.0-or-later */

#include "core/tensor_ops.hpp"
#include "core/cuda_memory_guard.hpp"
#include <hip/hip_runtime.h>
#include <>
#include <hipcub/hipcub.hpp>
#include <cfloat>
#include <limits>

// Thrust headers
#include <thrust/device_ptr.h>
#include <thrust/transform.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/gather.h>
#include <thrust/scatter.h>
#include <thrust/scan.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/tuple.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>

namespace gs::tensor_ops {

// ============= Import broadcast index calculator =============
// This function is defined in tensor_ops.cu
__device__ inline size_t compute_broadcast_index(
    size_t idx, const size_t* src_shape, size_t src_rank,
    const size_t* dst_shape, size_t dst_rank) {

    size_t src_idx = 0, dst_stride = 1;

    #pragma unroll 8
    for (int i = dst_rank - 1; i >= 0; --i) {
        size_t dst_coord = (idx / dst_stride) % dst_shape[i];
        int src_dim = i - (dst_rank - src_rank);

        if (src_dim >= 0) {
            size_t src_coord = (src_shape[src_dim] == 1) ? 0 : dst_coord;
            size_t src_stride = 1;
            for (int j = src_dim + 1; j < src_rank; ++j) {
                src_stride *= src_shape[j];
            }
            src_idx += src_coord * src_stride;
        }

        dst_stride *= dst_shape[i];
    }

    return src_idx;
}

// ============= Comparison Kernels (with broadcasting) =============
// These need custom kernels for broadcasting support
__global__ void compare_eq_kernel(const float* a, const float* b, unsigned char* c,
                                  const size_t* shapes, size_t info, size_t total) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total) return;

    size_t a_rank = info & 0x1F;
    size_t b_rank = (info >> 5) & 0x1F;
    size_t c_rank = (info >> 10) & 0x1F;
    bool fast_path = info & 0x8000;

    if (fast_path) {
        c[idx] = (a[idx] == b[idx]) ? 1 : 0;
    } else {
        size_t a_idx = compute_broadcast_index(idx, shapes, a_rank, shapes + 20, c_rank);
        size_t b_idx = compute_broadcast_index(idx, shapes + 10, b_rank, shapes + 20, c_rank);
        c[idx] = (a[a_idx] == b[b_idx]) ? 1 : 0;
    }
}

__global__ void compare_lt_kernel(const float* a, const float* b, unsigned char* c,
                                  const size_t* shapes, size_t info, size_t total) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total) return;

    size_t a_rank = info & 0x1F;
    size_t b_rank = (info >> 5) & 0x1F;
    size_t c_rank = (info >> 10) & 0x1F;
    bool fast_path = info & 0x8000;

    if (fast_path) {
        c[idx] = (a[idx] < b[idx]) ? 1 : 0;
    } else {
        size_t a_idx = compute_broadcast_index(idx, shapes, a_rank, shapes + 20, c_rank);
        size_t b_idx = compute_broadcast_index(idx, shapes + 10, b_rank, shapes + 20, c_rank);
        c[idx] = (a[a_idx] < b[b_idx]) ? 1 : 0;
    }
}

__global__ void compare_gt_kernel(const float* a, const float* b, unsigned char* c,
                                  const size_t* shapes, size_t info, size_t total) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total) return;

    size_t a_rank = info & 0x1F;
    size_t b_rank = (info >> 5) & 0x1F;
    size_t c_rank = (info >> 10) & 0x1F;
    bool fast_path = info & 0x8000;

    if (fast_path) {
        c[idx] = (a[idx] > b[idx]) ? 1 : 0;
    } else {
        size_t a_idx = compute_broadcast_index(idx, shapes, a_rank, shapes + 20, c_rank);
        size_t b_idx = compute_broadcast_index(idx, shapes + 10, b_rank, shapes + 20, c_rank);
        c[idx] = (a[a_idx] > b[b_idx]) ? 1 : 0;
    }
}

// ============= Scalar comparison functors =============
struct CompareScalarEqFunctor {
    float val;
    CompareScalarEqFunctor(float v) : val(v) {}
    __host__ __device__ unsigned char operator()(float x) const {
        return (x == val) ? 1 : 0;
    }
};

struct CompareScalarLtFunctor {
    float val;
    CompareScalarLtFunctor(float v) : val(v) {}
    __host__ __device__ unsigned char operator()(float x) const {
        return (x < val) ? 1 : 0;
    }
};

struct CompareScalarGtFunctor {
    float val;
    CompareScalarGtFunctor(float v) : val(v) {}
    __host__ __device__ unsigned char operator()(float x) const {
        return (x > val) ? 1 : 0;
    }
};

// Scalar comparison kernels using Thrust
void launch_compare_scalar_eq(const float* a, float val, unsigned char* r, size_t n, hipStream_t s) {
    auto a_ptr = thrust::device_pointer_cast(a);
    auto r_ptr = thrust::device_pointer_cast(r);

    thrust::transform(
        thrust::cuda::par.on(s),
        a_ptr, a_ptr + n,
        r_ptr,
        CompareScalarEqFunctor(val)
    );
}

void launch_compare_scalar_lt(const float* a, float val, unsigned char* r, size_t n, hipStream_t s) {
    auto a_ptr = thrust::device_pointer_cast(a);
    auto r_ptr = thrust::device_pointer_cast(r);

    thrust::transform(
        thrust::cuda::par.on(s),
        a_ptr, a_ptr + n,
        r_ptr,
        CompareScalarLtFunctor(val)
    );
}

void launch_compare_scalar_gt(const float* a, float val, unsigned char* r, size_t n, hipStream_t s) {
    auto a_ptr = thrust::device_pointer_cast(a);
    auto r_ptr = thrust::device_pointer_cast(r);

    thrust::transform(
        thrust::cuda::par.on(s),
        a_ptr, a_ptr + n,
        r_ptr,
        CompareScalarGtFunctor(val)
    );
}

// ============= Logical Operation Kernels (with broadcasting) =============
__global__ void logical_and_kernel(const unsigned char* a, const unsigned char* b,
                                   unsigned char* c, const size_t* shapes,
                                   size_t info, size_t total) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total) return;

    size_t a_rank = info & 0x1F;
    size_t b_rank = (info >> 5) & 0x1F;
    size_t c_rank = (info >> 10) & 0x1F;
    bool fast_path = info & 0x8000;

    if (fast_path) {
        c[idx] = (a[idx] && b[idx]) ? 1 : 0;
    } else {
        size_t a_idx = compute_broadcast_index(idx, shapes, a_rank, shapes + 20, c_rank);
        size_t b_idx = compute_broadcast_index(idx, shapes + 10, b_rank, shapes + 20, c_rank);
        c[idx] = (a[a_idx] && b[b_idx]) ? 1 : 0;
    }
}

__global__ void logical_or_kernel(const unsigned char* a, const unsigned char* b,
                                  unsigned char* c, const size_t* shapes,
                                  size_t info, size_t total) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total) return;

    size_t a_rank = info & 0x1F;
    size_t b_rank = (info >> 5) & 0x1F;
    size_t c_rank = (info >> 10) & 0x1F;
    bool fast_path = info & 0x8000;

    if (fast_path) {
        c[idx] = (a[idx] || b[idx]) ? 1 : 0;
    } else {
        size_t a_idx = compute_broadcast_index(idx, shapes, a_rank, shapes + 20, c_rank);
        size_t b_idx = compute_broadcast_index(idx, shapes + 10, b_rank, shapes + 20, c_rank);
        c[idx] = (a[a_idx] || b[b_idx]) ? 1 : 0;
    }
}

__global__ void logical_xor_kernel(const unsigned char* a, const unsigned char* b,
                                   unsigned char* c, const size_t* shapes,
                                   size_t info, size_t total) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= total) return;

    size_t a_rank = info & 0x1F;
    size_t b_rank = (info >> 5) & 0x1F;
    size_t c_rank = (info >> 10) & 0x1F;
    bool fast_path = info & 0x8000;

    if (fast_path) {
        c[idx] = ((a[idx] != 0) != (b[idx] != 0)) ? 1 : 0;
    } else {
        size_t a_idx = compute_broadcast_index(idx, shapes, a_rank, shapes + 20, c_rank);
        size_t b_idx = compute_broadcast_index(idx, shapes + 10, b_rank, shapes + 20, c_rank);
        c[idx] = ((a[a_idx] != 0) != (b[b_idx] != 0)) ? 1 : 0;
    }
}

// Logical NOT using Thrust
struct LogicalNotFunctor {
    __host__ __device__ unsigned char operator()(unsigned char x) const {
        return !x;
    }
};

void launch_logical_not(const unsigned char* a, unsigned char* r, size_t n, hipStream_t s) {
    auto a_ptr = thrust::device_pointer_cast(a);
    auto r_ptr = thrust::device_pointer_cast(r);

    thrust::transform(
        thrust::cuda::par.on(s),
        a_ptr, a_ptr + n,
        r_ptr,
        LogicalNotFunctor()
    );
}

// ============= Launch Functions for Comparison/Logical Ops =============
void launch_compare_eq(const float* a, const float* b, unsigned char* c,
                      const size_t* a_shape, const size_t* b_shape, const size_t* c_shape,
                      size_t a_rank, size_t b_rank, size_t c_rank,
                      size_t c_elements, hipStream_t stream) {
    static thread_local CudaDeviceMemory<size_t> shapes(30);

    size_t h_shapes[30] = {0};
    std::copy(a_shape, a_shape + a_rank, h_shapes);
    std::copy(b_shape, b_shape + b_rank, h_shapes + 10);
    std::copy(c_shape, c_shape + c_rank, h_shapes + 20);
    shapes.copy_from_host(h_shapes, 30);

    bool fast_path = (a_rank == c_rank && b_rank == c_rank &&
                     std::equal(a_shape, a_shape + a_rank, c_shape) &&
                     std::equal(b_shape, b_shape + b_rank, c_shape));
    size_t info = a_rank | (b_rank << 5) | (c_rank << 10) | (fast_path << 15);

    int blocks = (c_elements + 255) / 256;
    compare_eq_kernel<<<blocks, 256, 0, stream>>>(a, b, c, shapes.get(), info, c_elements);
}

void launch_compare_lt(const float* a, const float* b, unsigned char* c,
                      const size_t* a_shape, const size_t* b_shape, const size_t* c_shape,
                      size_t a_rank, size_t b_rank, size_t c_rank,
                      size_t c_elements, hipStream_t stream) {
    static thread_local CudaDeviceMemory<size_t> shapes(30);

    size_t h_shapes[30] = {0};
    std::copy(a_shape, a_shape + a_rank, h_shapes);
    std::copy(b_shape, b_shape + b_rank, h_shapes + 10);
    std::copy(c_shape, c_shape + c_rank, h_shapes + 20);
    shapes.copy_from_host(h_shapes, 30);

    bool fast_path = (a_rank == c_rank && b_rank == c_rank &&
                     std::equal(a_shape, a_shape + a_rank, c_shape) &&
                     std::equal(b_shape, b_shape + b_rank, c_shape));
    size_t info = a_rank | (b_rank << 5) | (c_rank << 10) | (fast_path << 15);

    int blocks = (c_elements + 255) / 256;
    compare_lt_kernel<<<blocks, 256, 0, stream>>>(a, b, c, shapes.get(), info, c_elements);
}

void launch_compare_gt(const float* a, const float* b, unsigned char* c,
                      const size_t* a_shape, const size_t* b_shape, const size_t* c_shape,
                      size_t a_rank, size_t b_rank, size_t c_rank,
                      size_t c_elements, hipStream_t stream) {
    static thread_local CudaDeviceMemory<size_t> shapes(30);

    size_t h_shapes[30] = {0};
    std::copy(a_shape, a_shape + a_rank, h_shapes);
    std::copy(b_shape, b_shape + b_rank, h_shapes + 10);
    std::copy(c_shape, c_shape + c_rank, h_shapes + 20);
    shapes.copy_from_host(h_shapes, 30);

    bool fast_path = (a_rank == c_rank && b_rank == c_rank &&
                     std::equal(a_shape, a_shape + a_rank, c_shape) &&
                     std::equal(b_shape, b_shape + b_rank, c_shape));
    size_t info = a_rank | (b_rank << 5) | (c_rank << 10) | (fast_path << 15);

    int blocks = (c_elements + 255) / 256;
    compare_gt_kernel<<<blocks, 256, 0, stream>>>(a, b, c, shapes.get(), info, c_elements);
}

void launch_logical_and(const unsigned char* a, const unsigned char* b, unsigned char* c,
                       const size_t* a_shape, const size_t* b_shape, const size_t* c_shape,
                       size_t a_rank, size_t b_rank, size_t c_rank,
                       size_t c_elements, hipStream_t stream) {
    static thread_local CudaDeviceMemory<size_t> shapes(30);

    size_t h_shapes[30] = {0};
    std::copy(a_shape, a_shape + a_rank, h_shapes);
    std::copy(b_shape, b_shape + b_rank, h_shapes + 10);
    std::copy(c_shape, c_shape + c_rank, h_shapes + 20);
    shapes.copy_from_host(h_shapes, 30);

    bool fast_path = (a_rank == c_rank && b_rank == c_rank &&
                     std::equal(a_shape, a_shape + a_rank, c_shape) &&
                     std::equal(b_shape, b_shape + b_rank, c_shape));
    size_t info = a_rank | (b_rank << 5) | (c_rank << 10) | (fast_path << 15);

    int blocks = (c_elements + 255) / 256;
    logical_and_kernel<<<blocks, 256, 0, stream>>>(a, b, c, shapes.get(), info, c_elements);
}

void launch_logical_or(const unsigned char* a, const unsigned char* b, unsigned char* c,
                      const size_t* a_shape, const size_t* b_shape, const size_t* c_shape,
                      size_t a_rank, size_t b_rank, size_t c_rank,
                      size_t c_elements, hipStream_t stream) {
    static thread_local CudaDeviceMemory<size_t> shapes(30);

    size_t h_shapes[30] = {0};
    std::copy(a_shape, a_shape + a_rank, h_shapes);
    std::copy(b_shape, b_shape + b_rank, h_shapes + 10);
    std::copy(c_shape, c_shape + c_rank, h_shapes + 20);
    shapes.copy_from_host(h_shapes, 30);

    bool fast_path = (a_rank == c_rank && b_rank == c_rank &&
                     std::equal(a_shape, a_shape + a_rank, c_shape) &&
                     std::equal(b_shape, b_shape + b_rank, c_shape));
    size_t info = a_rank | (b_rank << 5) | (c_rank << 10) | (fast_path << 15);

    int blocks = (c_elements + 255) / 256;
    logical_or_kernel<<<blocks, 256, 0, stream>>>(a, b, c, shapes.get(), info, c_elements);
}

void launch_logical_xor(const unsigned char* a, const unsigned char* b, unsigned char* c,
                       const size_t* a_shape, const size_t* b_shape, const size_t* c_shape,
                       size_t a_rank, size_t b_rank, size_t c_rank,
                       size_t c_elements, hipStream_t stream) {
    static thread_local CudaDeviceMemory<size_t> shapes(30);

    size_t h_shapes[30] = {0};
    std::copy(a_shape, a_shape + a_rank, h_shapes);
    std::copy(b_shape, b_shape + b_rank, h_shapes + 10);
    std::copy(c_shape, c_shape + c_rank, h_shapes + 20);
    shapes.copy_from_host(h_shapes, 30);

    bool fast_path = (a_rank == c_rank && b_rank == c_rank &&
                     std::equal(a_shape, a_shape + a_rank, c_shape) &&
                     std::equal(b_shape, b_shape + b_rank, c_shape));
    size_t info = a_rank | (b_rank << 5) | (c_rank << 10) | (fast_path << 15);

    int blocks = (c_elements + 255) / 256;
    logical_xor_kernel<<<blocks, 256, 0, stream>>>(a, b, c, shapes.get(), info, c_elements);
}

// ============= Masking Operations using Thrust =============

// Masked fill using Thrust
struct MaskedFillFunctor {
    float val;
    MaskedFillFunctor(float v) : val(v) {}

    __host__ __device__ float operator()(const thrust::tuple<float, unsigned char>& t) const {
        return thrust::get<1>(t) ? val : thrust::get<0>(t);
    }
};

void launch_masked_fill(float* data, const unsigned char* mask, float val, size_t n, hipStream_t s) {
    auto data_ptr = thrust::device_pointer_cast(data);
    auto mask_ptr = thrust::device_pointer_cast(mask);

    auto begin = thrust::make_zip_iterator(thrust::make_tuple(data_ptr, mask_ptr));
    auto end = thrust::make_zip_iterator(thrust::make_tuple(data_ptr + n, mask_ptr + n));

    thrust::transform(
        thrust::cuda::par.on(s),
        begin, end,
        data_ptr,
        MaskedFillFunctor(val)
    );
}

// Masked select using Thrust copy_if
struct ExtractValueFunctor {
    __host__ __device__ float operator()(const thrust::tuple<float, unsigned char>& t) const {
        return thrust::get<0>(t);
    }
};

struct ExtractMaskFunctor {
    __host__ __device__ bool operator()(const thrust::tuple<float, unsigned char>& t) const {
        return thrust::get<1>(t) != 0;
    }
};

void launch_masked_select(const float* input, const unsigned char* mask,
                         float* output, size_t n, size_t output_size, hipStream_t stream) {
    if (n == 0 || output_size == 0) return;

    auto input_ptr = thrust::device_pointer_cast(input);
    auto mask_ptr = thrust::device_pointer_cast(mask);
    auto output_ptr = thrust::device_pointer_cast(output);

    auto begin = thrust::make_zip_iterator(thrust::make_tuple(input_ptr, mask_ptr));
    auto end = thrust::make_zip_iterator(thrust::make_tuple(input_ptr + n, mask_ptr + n));

    auto transform_begin = thrust::make_transform_iterator(begin, ExtractValueFunctor());
    auto transform_end = thrust::make_transform_iterator(end, ExtractValueFunctor());
    auto mask_begin = thrust::make_transform_iterator(begin, ExtractMaskFunctor());

    thrust::copy_if(
        thrust::cuda::par.on(stream),
        transform_begin, transform_end,
        mask_begin,
        output_ptr,
        thrust::identity<bool>()
    );
}

// Masked scatter using CUB scan (keep existing implementation as it's optimal)
__global__ void masked_scatter_compact_kernel(float* data, const unsigned char* mask,
                                             const float* src, const int* scan, size_t n) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n && mask[idx]) {
        data[idx] = src[scan[idx]];
    }
}

void launch_masked_scatter(float* data, const unsigned char* mask,
                          const float* src, size_t n, size_t src_size, hipStream_t stream) {
    if (n == 0 || src_size == 0) return;

    CudaDeviceMemory<int> scan_result(n);
    if (!scan_result.valid()) return;

    size_t temp_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(nullptr, temp_bytes, mask, scan_result.get(), n, stream);

    CudaDeviceMemory<uint8_t> temp_storage(temp_bytes);
    if (!temp_storage.valid()) return;

    hipcub::DeviceScan::ExclusiveSum(temp_storage.get(), temp_bytes,
                                  mask, scan_result.get(), n, stream);

    if (stream != 0) hipStreamSynchronize(stream);

    int blocks = (n + 255) / 256;
    masked_scatter_compact_kernel<<<blocks, 256, 0, stream>>>(
        data, mask, src, scan_result.get(), n);
}

// ============= Where Operation with Broadcasting =============
__global__ void where_kernel(const unsigned char* cond, const float* x, const float* y,
                            float* r, const size_t* shapes,
                            size_t cr, size_t xr, size_t yr, size_t rr, size_t n) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    size_t c_idx = compute_broadcast_index(idx, shapes, cr, shapes + 30, rr);
    size_t x_idx = compute_broadcast_index(idx, shapes + 10, xr, shapes + 30, rr);
    size_t y_idx = compute_broadcast_index(idx, shapes + 20, yr, shapes + 30, rr);

    r[idx] = cond[c_idx] ? x[x_idx] : y[y_idx];
}

void launch_where(const unsigned char* cond, const float* x, const float* y, float* r,
                 const size_t* cond_shape, const size_t* x_shape,
                 const size_t* y_shape, const size_t* r_shape,
                 size_t cond_rank, size_t x_rank, size_t y_rank, size_t r_rank,
                 size_t total, hipStream_t stream) {

    static thread_local CudaDeviceMemory<size_t> shapes(40);

    size_t h_shapes[40] = {0};
    std::copy(cond_shape, cond_shape + cond_rank, h_shapes);
    std::copy(x_shape, x_shape + x_rank, h_shapes + 10);
    std::copy(y_shape, y_shape + y_rank, h_shapes + 20);
    std::copy(r_shape, r_shape + r_rank, h_shapes + 30);
    shapes.copy_from_host(h_shapes, 40);

    where_kernel<<<(total + 255) / 256, 256, 0, stream>>>(
        cond, x, y, r, shapes.get(), cond_rank, x_rank, y_rank, r_rank, total);
}

// ============= Count Nonzero using Thrust =============
struct IsNonzeroFloat {
    __host__ __device__ bool operator()(float x) const {
        return x != 0.0f;
    }
};

struct IsNonzeroBool {
    __host__ __device__ bool operator()(unsigned char x) const {
        return x != 0;
    }
};

void launch_count_nonzero_bool(const unsigned char* data, size_t* count,
                              size_t n, hipStream_t stream) {
    auto data_ptr = thrust::device_pointer_cast(data);

    // Thrust count_if returns the result directly to host - no need to copy!
    *count = thrust::count_if(
        thrust::cuda::par.on(stream),
        data_ptr, data_ptr + n,
        IsNonzeroBool()
    );
}

void launch_count_nonzero_float(const float* data, size_t* count,
                               size_t n, hipStream_t stream) {
    auto data_ptr = thrust::device_pointer_cast(data);

    // Thrust count_if returns the result directly to host - no need to copy!
    *count = thrust::count_if(
        thrust::cuda::par.on(stream),
        data_ptr, data_ptr + n,
        IsNonzeroFloat()
    );
}

// ============= Index Operations =============

// Index select - keep custom kernel for boundary modes
__global__ void index_select_kernel(const float* in, const int* idx, float* out,
                                   size_t outer, size_t dim_size, size_t inner,
                                   size_t idx_size, int boundary) {
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = outer * idx_size * inner;

    if (tid >= total) return;

    size_t o = tid / (idx_size * inner);
    size_t i = (tid / inner) % idx_size;
    size_t j = tid % inner;

    int sel = idx[i];

    if (boundary == 1) sel = max(0, min((int)dim_size - 1, sel));
    else if (boundary == 2) sel = ((sel % (int)dim_size) + dim_size) % dim_size;
    else if (sel < 0 || sel >= dim_size) { out[tid] = 0; return; }

    out[tid] = in[o * dim_size * inner + sel * inner + j];
}

void launch_index_select(const float* in, const int* idx, float* out,
                        const size_t* shape, size_t rank, int dim,
                        size_t idx_size, int boundary, hipStream_t stream) {
    size_t outer = 1, inner = 1;
    for (int i = 0; i < dim; ++i) outer *= shape[i];
    for (size_t i = dim + 1; i < rank; ++i) inner *= shape[i];

    size_t total = outer * idx_size * inner;
    index_select_kernel<<<(total + 255) / 256, 256, 0, stream>>>(
        in, idx, out, outer, shape[dim], inner, idx_size, boundary);
}

// Gather - keep custom kernel for complex indexing
__global__ void gather_kernel(const float* in, const int* idx, float* out,
                              const size_t* in_shape, const size_t* idx_shape,
                              size_t in_rank, size_t idx_rank, int dim, size_t total, int boundary) {
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= total) return;

    size_t in_strides[10];
    in_strides[in_rank - 1] = 1;
    for (int i = in_rank - 2; i >= 0; --i) {
        in_strides[i] = in_strides[i + 1] * in_shape[i + 1];
    }

    size_t out_strides[10];
    out_strides[idx_rank - 1] = 1;
    for (int i = idx_rank - 2; i >= 0; --i) {
        out_strides[i] = out_strides[i + 1] * idx_shape[i + 1];
    }

    size_t out_coords[10] = {0};
    size_t temp = tid;
    for (size_t d = 0; d < idx_rank; ++d) {
        out_coords[d] = temp / out_strides[d];
        temp %= out_strides[d];
    }

    int gather_idx = idx[tid];

    if (boundary == 1) {
        gather_idx = max(0, min((int)in_shape[dim] - 1, gather_idx));
    } else if (boundary == 2) {
        gather_idx = ((gather_idx % (int)in_shape[dim]) + in_shape[dim]) % in_shape[dim];
    } else if (gather_idx < 0 || gather_idx >= in_shape[dim]) {
        out[tid] = 0;
        return;
    }

    size_t src_idx = 0;
    for (size_t d = 0; d < in_rank; ++d) {
        size_t coord;
        if (d == dim) {
            coord = gather_idx;
        } else if (d < idx_rank) {
            coord = out_coords[d];
        } else {
            coord = 0;
        }

        if (coord >= in_shape[d]) {
            out[tid] = 0;
            return;
        }

        src_idx += coord * in_strides[d];
    }

    out[tid] = in[src_idx];
}

void launch_gather(const float* in, const int* idx, float* out,
                  const size_t* in_shape, const size_t* idx_shape,
                  size_t rank, int dim, size_t total, int boundary, hipStream_t stream) {
    CudaDeviceMemory<size_t> d_in_shape(10);
    CudaDeviceMemory<size_t> d_idx_shape(10);

    size_t h_in_shape[10] = {0};
    size_t h_idx_shape[10] = {0};

    size_t idx_rank = rank;
    size_t idx_elements = 1;
    for (size_t i = 0; i < rank; ++i) {
        if (idx_shape[i] > 0) {
            h_idx_shape[i] = idx_shape[i];
            idx_elements *= idx_shape[i];
        } else {
            break;
        }
    }

    idx_rank = 0;
    size_t check_elements = 1;
    for (size_t i = 0; i < 10; ++i) {
        if (idx_shape[i] > 0) {
            check_elements *= idx_shape[i];
            idx_rank++;
            if (check_elements == total) break;
        } else {
            break;
        }
    }

    if (idx_rank == 0) idx_rank = 1;

    for (size_t i = 0; i < rank; ++i) {
        h_in_shape[i] = in_shape[i];
    }

    d_in_shape.copy_from_host(h_in_shape, 10);
    d_idx_shape.copy_from_host(h_idx_shape, 10);

    int blocks = (total + 255) / 256;
    gather_kernel<<<blocks, 256, 0, stream>>>(
        in, idx, out, d_in_shape.get(), d_idx_shape.get(),
        rank, idx_rank, dim, total, boundary);
}

// Take using Thrust gather
struct IndexClampFunctor {
    size_t size;
    IndexClampFunctor(size_t s) : size(s) {}

    __host__ __device__ size_t operator()(int idx) const {
        if (idx < 0) idx += size;
        if (idx < 0 || idx >= size) return 0;
        return idx;
    }
};

void launch_take(const float* in, const int* idx, float* out,
                size_t in_size, size_t out_size, hipStream_t stream) {
    auto in_ptr = thrust::device_pointer_cast(in);
    auto idx_ptr = thrust::device_pointer_cast(idx);
    auto out_ptr = thrust::device_pointer_cast(out);

    // Create transform iterator to handle negative indices
    auto transform_idx = thrust::make_transform_iterator(idx_ptr, IndexClampFunctor(in_size));

    thrust::gather(
        thrust::cuda::par.on(stream),
        transform_idx, transform_idx + out_size,
        in_ptr,
        out_ptr
    );
}

// Scatter operations
__global__ void scatter_kernel(float* out, const int* idx, const float* in,
                              size_t outer, size_t dim_sz, size_t inner,
                              size_t idx_sz, int mode) {
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    size_t n = outer * idx_sz * inner;
    if (tid >= n) return;

    size_t outer_idx = tid / (idx_sz * inner);
    size_t idx_pos = (tid / inner) % idx_sz;
    size_t inner_idx = tid % inner;

    int scatter_idx = idx[idx_pos];
    if (scatter_idx < 0 || scatter_idx >= dim_sz) return;

    size_t dst_idx = outer_idx * dim_sz * inner + scatter_idx * inner + inner_idx;

    if (mode == 1) {
        atomicAdd(&out[dst_idx], in[tid]);
    } else {
        out[dst_idx] = in[tid];
    }
}

void launch_scatter(float* out, const int* idx, const float* in,
                   const size_t* out_shape, const size_t* in_shape,
                   size_t rank, int dim, size_t total, int mode, hipStream_t stream) {
    size_t outer = 1, inner = 1;
    for (int i = 0; i < dim; ++i) outer *= out_shape[i];
    for (size_t i = dim + 1; i < rank; ++i) inner *= out_shape[i];

    scatter_kernel<<<(total + 255) / 256, 256, 0, stream>>>(
        out, idx, in, outer, out_shape[dim], inner, in_shape[dim], mode);
}

void launch_index_fill(float* data, const int* idx, float val,
                      const size_t* shape, size_t rank, int dim,
                      size_t n_idx, hipStream_t stream) {
    CudaDeviceMemory<float> val_buffer(n_idx);

    auto val_ptr = thrust::device_pointer_cast(val_buffer.get());
    thrust::fill(thrust::cuda::par.on(stream), val_ptr, val_ptr + n_idx, val);

    size_t in_shape[10] = {0};
    std::copy(shape, shape + rank, in_shape);
    in_shape[dim] = n_idx;

    launch_scatter(data, idx, val_buffer.get(), shape, in_shape, rank, dim, n_idx, 0, stream);
}

void launch_index_copy(float* dst, const int* idx, const float* src,
                      const size_t* shape, size_t rank, int dim,
                      size_t n_idx, hipStream_t stream) {
    size_t in_shape[10] = {0};
    std::copy(shape, shape + rank, in_shape);
    in_shape[dim] = n_idx;

    launch_scatter(dst, idx, src, shape, in_shape, rank, dim, n_idx, 0, stream);
}

void launch_index_add(float* dst, const int* idx, const float* src,
                     const size_t* shape, size_t rank, int dim,
                     size_t n_idx, hipStream_t stream) {
    size_t in_shape[10] = {0};
    std::copy(shape, shape + rank, in_shape);
    in_shape[dim] = n_idx;

    launch_scatter(dst, idx, src, shape, in_shape, rank, dim, n_idx, 1, stream);
}

// Index put using Thrust scatter
void launch_index_put(float* data, const int* idx, const float* vals,
                     size_t data_size, size_t idx_size, hipStream_t stream) {
    auto data_ptr = thrust::device_pointer_cast(data);
    auto idx_ptr = thrust::device_pointer_cast(idx);
    auto vals_ptr = thrust::device_pointer_cast(vals);

    // Transform indices to handle negative values
    auto transform_idx = thrust::make_transform_iterator(idx_ptr, IndexClampFunctor(data_size));

    thrust::scatter(
        thrust::cuda::par.on(stream),
        vals_ptr, vals_ptr + idx_size,
        transform_idx,
        data_ptr
    );
}

// ============= Nonzero Operations using Thrust =============

struct NonzeroFloat {
    __host__ __device__ bool operator()(float x) const {
        return x != 0.0f;
    }
};

struct NonzeroBool {
    __host__ __device__ bool operator()(unsigned char x) const {
        return x != 0;
    }
};

void launch_nonzero(const float* data, int64_t* indices,
                   size_t n, size_t output_size, hipStream_t stream) {
    if (n == 0 || output_size == 0) return;

    auto data_ptr = thrust::device_pointer_cast(data);
    auto indices_ptr = thrust::device_pointer_cast(indices);
    auto counting = thrust::counting_iterator<int64_t>(0);

    // Copy indices where data is nonzero
    thrust::copy_if(
        thrust::cuda::par.on(stream),
        counting, counting + n,
        data_ptr,
        indices_ptr,
        NonzeroFloat()
    );
}

void launch_nonzero_bool(const unsigned char* data, int64_t* indices,
                         size_t n, size_t output_size, hipStream_t stream) {
    if (n == 0 || output_size == 0) return;

    auto data_ptr = thrust::device_pointer_cast(data);
    auto indices_ptr = thrust::device_pointer_cast(indices);
    auto counting = thrust::counting_iterator<int64_t>(0);

    // Copy indices where data is nonzero
    thrust::copy_if(
        thrust::cuda::par.on(stream),
        counting, counting + n,
        data_ptr,
        indices_ptr,
        NonzeroBool()
    );
}

} // namespace gs::tensor_ops