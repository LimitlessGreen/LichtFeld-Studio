#include "hip/hip_runtime.h"
/* SPDX-FileCopyrightText: 2025 LichtFeld Studio Authors
 * SPDX-License-Identifier: GPL-3.0-or-later */

#include "internal/tensor_ops.hpp"
#include <hipblas.h>
#include <hip/hip_runtime.h>

namespace gs::tensor_ops {

    // ============= Matrix Operations Kernels =============

    // Optimized transpose kernel using shared memory
    template <int TILE_DIM, int BLOCK_ROWS>
    __global__ void transpose_kernel(const float* input, float* output,
                                     size_t rows, size_t cols) {
        __shared__ float tile[TILE_DIM][TILE_DIM + 1]; // +1 to avoid bank conflicts

        int x = blockIdx.x * TILE_DIM + threadIdx.x;
        int y = blockIdx.y * TILE_DIM + threadIdx.y;
        int width = cols;

        for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
            if (x < cols && (y + j) < rows) {
                tile[threadIdx.y + j][threadIdx.x] = input[(y + j) * width + x];
            }
        }

        __syncthreads();

        x = blockIdx.y * TILE_DIM + threadIdx.x;
        y = blockIdx.x * TILE_DIM + threadIdx.y;
        width = rows;

        for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
            if (x < rows && (y + j) < cols) {
                output[(y + j) * width + x] = tile[threadIdx.x][threadIdx.y + j];
            }
        }
    }

    // Simple matrix multiply for batch operations (when cuBLAS batch isn't available)
    __global__ void batch_matmul_kernel(const float* a, const float* b, float* c,
                                        size_t batch_size, size_t m, size_t n, size_t k) {
        int batch_idx = blockIdx.z;
        if (batch_idx >= batch_size)
            return;

        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;

        if (row < m && col < n) {
            float sum = 0.0f;

            const float* a_batch = a + batch_idx * m * k;
            const float* b_batch = b + batch_idx * k * n;
            float* c_batch = c + batch_idx * m * n;

            for (size_t i = 0; i < k; ++i) {
                sum += a_batch[row * k + i] * b_batch[i * n + col];
            }

            c_batch[row * n + col] = sum;
        }
    }

    // Eye matrix creation
    __global__ void eye_kernel(float* data, size_t m, size_t n) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        size_t total = m * n;

        if (idx < total) {
            int row = idx / n;
            int col = idx % n;
            data[idx] = (row == col) ? 1.0f : 0.0f;
        }
    }

    // Create diagonal matrix from vector
    __global__ void diag_kernel(const float* diagonal, float* matrix, size_t n) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        size_t total = n * n;

        if (idx < total) {
            int row = idx / n;
            int col = idx % n;
            matrix[idx] = (row == col) ? diagonal[row] : 0.0f;
        }
    }

    // Extract diagonal from matrix
    __global__ void extract_diag_kernel(const float* matrix, float* diagonal, size_t n) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < n) {
            diagonal[idx] = matrix[idx * n + idx];
        }
    }

    // ============= Launch Functions Implementation =============
    void launch_eye(float* data, size_t m, size_t n, hipStream_t stream) {
        size_t total = m * n;
        int block_size = 256;
        int grid_size = (total + block_size - 1) / block_size;
        eye_kernel<<<grid_size, block_size, 0, stream>>>(data, m, n);
    }

    void launch_diag(const float* diagonal, float* matrix, size_t n, hipStream_t stream) {
        size_t total = n * n;
        int block_size = 256;
        int grid_size = (total + block_size - 1) / block_size;
        diag_kernel<<<grid_size, block_size, 0, stream>>>(diagonal, matrix, n);
    }

    void launch_extract_diag(const float* matrix, float* diagonal, size_t n, hipStream_t stream) {
        int block_size = 256;
        int grid_size = (n + block_size - 1) / block_size;
        extract_diag_kernel<<<grid_size, block_size, 0, stream>>>(matrix, diagonal, n);
    }

} // namespace gs::tensor_ops