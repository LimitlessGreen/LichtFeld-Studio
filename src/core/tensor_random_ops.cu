#include "hip/hip_runtime.h"
/* SPDX-FileCopyrightText: 2025 LichtFeld Studio Authors
 * SPDX-License-Identifier: GPL-3.0-or-later */

#include "core/tensor_ops.hpp"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

// Thrust headers for multinomial without replacement
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

namespace gs::tensor_ops {

    // ============= Random Operations Kernels =============

    // Uniform random generation
    __global__ void uniform_kernel(float* data, size_t n, float low, float high,
                                   unsigned long long seed) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < n) {
            hiprandState state;
            // Use different seed per thread by adding thread index to base seed
            // This ensures each thread gets a different random sequence
            hiprand_init(seed + idx, 0, 0, &state);
            float val = hiprand_uniform(&state);
            data[idx] = val * (high - low) + low;
        }
    }

    // Normal random generation
    __global__ void normal_kernel(float* data, size_t n, float mean, float std,
                                  unsigned long long seed) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < n) {
            hiprandState state;
            // Use different seed per thread by adding thread index to base seed
            hiprand_init(seed + idx, 0, 0, &state);
            data[idx] = hiprand_normal(&state) * std + mean;
        }
    }

    // Bernoulli random generation
    __global__ void bernoulli_kernel(float* data, size_t n, float p,
                                     unsigned long long seed) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < n) {
            hiprandState state;
            // Use different seed per thread by adding thread index to base seed
            hiprand_init(seed + idx, 0, 0, &state);
            float val = hiprand_uniform(&state);
            data[idx] = (val < p) ? 1.0f : 0.0f;
        }
    }

    __global__ void randint_kernel(int* data, size_t n, int low, int high,
                                   unsigned long long seed) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < n) {
            hiprandState state;
            hiprand_init(seed + idx, 0, 0, &state);

            // Generate uniform [0, 1) and scale to [low, high)
            float val = hiprand_uniform(&state);
            int range = high - low;

            // Properly scale to integer range [low, high)
            int result = low + static_cast<int>(val * range);

            // Ensure we're within bounds
            if (result >= high) {
                result = high - 1;
            }
            if (result < low) {
                result = low;
            }

            data[idx] = result;
        }
    }

    // Kernel for multinomial sampling with replacement
    __global__ void multinomial_with_replacement_kernel(const float* weights, int* samples,
                                                        unsigned long n, unsigned long num_samples,
                                                        float sum, unsigned long long seed) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx >= num_samples) return;

        hiprandState state;
        hiprand_init(seed + idx, 0, 0, &state);

        // Generate random value in [0, sum)
        float u = hiprand_uniform(&state) * sum;

        // Find the index using linear search
        float cumsum = 0.0f;
        for (unsigned long i = 0; i < n; ++i) {
            cumsum += weights[i];
            if (u <= cumsum) {
                samples[idx] = i;
                return;
            }
        }

        // Fallback (shouldn't happen)
        samples[idx] = n - 1;
    }

    // Kernel to generate random keys for each index (Gumbel-max trick)
    __global__ void generate_gumbel_keys_kernel(const float* weights, float* keys,
                                                unsigned long n, unsigned long long seed) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx >= n) return;

        hiprandState state;
        hiprand_init(seed + idx, 0, 0, &state);

        // Generate Gumbel noise: -log(-log(uniform))
        float u = hiprand_uniform(&state);
        // Clamp to avoid log(0)
        u = fmaxf(u, 1e-10f);
        u = fminf(u, 1.0f - 1e-10f);

        float gumbel = -logf(-logf(u));

        // Add log-weight
        float log_weight = logf(fmaxf(weights[idx], 1e-10f));
        keys[idx] = log_weight + gumbel;
    }

    // ============= Launch Functions =============

    void launch_uniform(float* data, size_t n, float low, float high,
                        unsigned long long seed, hipStream_t stream) {
        int block_size = 256;
        int grid_size = (n + block_size - 1) / block_size;
        uniform_kernel<<<grid_size, block_size, 0, stream>>>(data, n, low, high, seed);
    }

    void launch_normal(float* data, size_t n, float mean, float std,
                       unsigned long long seed, hipStream_t stream) {
        int block_size = 256;
        int grid_size = (n + block_size - 1) / block_size;
        normal_kernel<<<grid_size, block_size, 0, stream>>>(data, n, mean, std, seed);
    }

    void launch_bernoulli(float* data, size_t n, float p,
                          unsigned long long seed, hipStream_t stream) {
        int block_size = 256;
        int grid_size = (n + block_size - 1) / block_size;
        bernoulli_kernel<<<grid_size, block_size, 0, stream>>>(data, n, p, seed);
    }

    void launch_randint(int* data, size_t n, int low, int high,
                        unsigned long long seed, hipStream_t stream) {
        int block_size = 256;
        int grid_size = (n + block_size - 1) / block_size;
        randint_kernel<<<grid_size, block_size, 0, stream>>>(data, n, low, high, seed);
    }

    void launch_multinomial(const float* weights, int* samples,
                             unsigned long n, unsigned long num_samples, bool replacement,
                             unsigned long long seed, hipStream_t stream) {
        if (n == 0 || num_samples == 0) return;

        // First, compute sum of weights using Thrust
        auto weights_ptr = thrust::device_pointer_cast(weights);
        float sum = thrust::reduce(
            thrust::cuda::par.on(stream),
            weights_ptr, weights_ptr + n,
            0.0f,
            thrust::plus<float>()
        );

        if (sum <= 0) {
            // Invalid weights, fill with zeros
            hipMemsetAsync(samples, 0, num_samples * sizeof(int), stream);
            return;
        }

        if (replacement) {
            // With replacement: simple multinomial sampling
            int block_size = 256;
            int grid_size = (num_samples + block_size - 1) / block_size;
            multinomial_with_replacement_kernel<<<grid_size, block_size, 0, stream>>>(
                weights, samples, n, num_samples, sum, seed);
        } else {
            // Without replacement: use Gumbel-max trick
            // 1. Generate Gumbel keys for each index
            // 2. Sort indices by keys (descending)
            // 3. Take first num_samples indices

            // Allocate temporary storage for keys and indices
            thrust::device_vector<float> keys(n);
            thrust::device_vector<int> indices(n);

            // Generate keys
            int block_size = 256;
            int grid_size = (n + block_size - 1) / block_size;
            generate_gumbel_keys_kernel<<<grid_size, block_size, 0, stream>>>(
                weights, thrust::raw_pointer_cast(keys.data()), n, seed);

            // Initialize indices [0, 1, 2, ..., n-1]
            thrust::sequence(
                thrust::cuda::par.on(stream),
                indices.begin(), indices.end()
            );

            // Sort indices by keys (descending order)
            thrust::sort_by_key(
                thrust::cuda::par.on(stream),
                keys.begin(), keys.end(),
                indices.begin(),
                thrust::greater<float>()
            );

            // Copy first num_samples indices to output
            // (num_samples is already capped by Tensor::multinomial for without replacement)
            thrust::copy_n(
                thrust::cuda::par.on(stream),
                indices.begin(),
                num_samples,
                thrust::device_pointer_cast(samples)
            );
        }
    }

} // namespace gs::tensor_ops