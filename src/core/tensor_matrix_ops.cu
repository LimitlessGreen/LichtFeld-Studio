#include "hip/hip_runtime.h"
/* SPDX-FileCopyrightText: 2025 LichtFeld Studio Authors
 * SPDX-License-Identifier: GPL-3.0-or-later */

#include "core/tensor_ops.hpp"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

namespace gs::tensor_ops {

    // ============= Matrix Operations Kernels =============

    // Optimized transpose kernel using shared memory
    template <int TILE_DIM, int BLOCK_ROWS>
    __global__ void transpose_kernel(const float* input, float* output,
                                     size_t rows, size_t cols) {
        __shared__ float tile[TILE_DIM][TILE_DIM + 1]; // +1 to avoid bank conflicts

        int x = blockIdx.x * TILE_DIM + threadIdx.x;
        int y = blockIdx.y * TILE_DIM + threadIdx.y;
        int width = cols;

        for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
            if (x < cols && (y + j) < rows) {
                tile[threadIdx.y + j][threadIdx.x] = input[(y + j) * width + x];
            }
        }

        __syncthreads();

        x = blockIdx.y * TILE_DIM + threadIdx.x;
        y = blockIdx.x * TILE_DIM + threadIdx.y;
        width = rows;

        for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
            if (x < rows && (y + j) < cols) {
                output[(y + j) * width + x] = tile[threadIdx.x][threadIdx.y + j];
            }
        }
    }

    // Simple matrix multiply for batch operations (when cuBLAS batch isn't available)
    __global__ void batch_matmul_kernel(const float* a, const float* b, float* c,
                                        size_t batch_size, size_t m, size_t n, size_t k) {
        int batch_idx = blockIdx.z;
        if (batch_idx >= batch_size)
            return;

        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;

        if (row < m && col < n) {
            float sum = 0.0f;

            const float* a_batch = a + batch_idx * m * k;
            const float* b_batch = b + batch_idx * k * n;
            float* c_batch = c + batch_idx * m * n;

            for (size_t i = 0; i < k; ++i) {
                sum += a_batch[row * k + i] * b_batch[i * n + col];
            }

            c_batch[row * n + col] = sum;
        }
    }

    // Eye matrix creation
    __global__ void eye_kernel(float* data, size_t m, size_t n) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        size_t total = m * n;

        if (idx < total) {
            int row = idx / n;
            int col = idx % n;
            data[idx] = (row == col) ? 1.0f : 0.0f;
        }
    }

    // Create diagonal matrix from vector
    __global__ void diag_kernel(const float* diagonal, float* matrix, size_t n) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;
        size_t total = n * n;

        if (idx < total) {
            int row = idx / n;
            int col = idx % n;
            matrix[idx] = (row == col) ? diagonal[row] : 0.0f;
        }
    }

    // Extract diagonal from matrix
    __global__ void extract_diag_kernel(const float* matrix, float* diagonal, size_t n) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < n) {
            diagonal[idx] = matrix[idx * n + idx];
        }
    }

    // ============= Random Operations Kernels =============

    // Uniform random generation
    __global__ void uniform_kernel(float* data, size_t n, float low, float high,
                                   unsigned long long seed) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < n) {
            hiprandState state;
            hiprand_init(seed, idx, 0, &state);
            float val = hiprand_uniform(&state);
            data[idx] = val * (high - low) + low;
        }
    }

    // Normal random generation
    __global__ void normal_kernel(float* data, size_t n, float mean, float std,
                                  unsigned long long seed) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < n) {
            hiprandState state;
            hiprand_init(seed, idx, 0, &state);
            data[idx] = hiprand_normal(&state) * std + mean;
        }
    }

    // Bernoulli random generation
    __global__ void bernoulli_kernel(float* data, size_t n, float p,
                                     unsigned long long seed) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < n) {
            hiprandState state;
            hiprand_init(seed, idx, 0, &state);
            float val = hiprand_uniform(&state);
            data[idx] = (val < p) ? 1.0f : 0.0f;
        }
    }

    // Random integer generation
    __global__ void randint_kernel(int* data, size_t n, int low, int high,
                                   unsigned long long seed) {
        int idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx < n) {
            hiprandState state;
            hiprand_init(seed, idx, 0, &state);
            // Generate uniform [0, 1) and scale to [low, high)
            float val = hiprand_uniform(&state);
            data[idx] = static_cast<int>(val * (high - low)) + low;
        }
    }

    // ============= Launch Functions =============

    void launch_matmul(const float* a, const float* b, float* c,
                       size_t m, size_t n, size_t k,
                       hipStream_t stream) {
        // For simple matrix multiply, use cuBLAS (handled in tensor_matrix_ops.cpp)
        // This is a fallback kernel if needed
        dim3 block(16, 16);
        dim3 grid((n + block.x - 1) / block.x, (m + block.y - 1) / block.y);

        // Simple naive kernel (cuBLAS is preferred)
        auto kernel = [=] __device__(int row, int col) {
            if (row < m && col < n) {
                float sum = 0.0f;
                for (size_t i = 0; i < k; ++i) {
                    sum += a[row * k + i] * b[i * n + col];
                }
                c[row * n + col] = sum;
            }
        };

        // For actual implementation, use cuBLAS in tensor_matrix_ops.cpp
    }

    void launch_batch_matmul(const float* a, const float* b, float* c,
                             size_t batch_size, size_t m, size_t n, size_t k,
                             hipStream_t stream) {
        dim3 block(16, 16);
        dim3 grid((n + block.x - 1) / block.x,
                  (m + block.y - 1) / block.y,
                  batch_size);

        batch_matmul_kernel<<<grid, block, 0, stream>>>(a, b, c, batch_size, m, n, k);
    }

    void launch_transpose(const float* input, float* output,
                          size_t rows, size_t cols,
                          hipStream_t stream) {
        const int TILE_DIM = 32;
        const int BLOCK_ROWS = 8;

        dim3 block(TILE_DIM, BLOCK_ROWS);
        dim3 grid((cols + TILE_DIM - 1) / TILE_DIM,
                  (rows + TILE_DIM - 1) / TILE_DIM);

        transpose_kernel<TILE_DIM, BLOCK_ROWS><<<grid, block, 0, stream>>>(
            input, output, rows, cols);
    }

    void launch_dot_product(const float* a, const float* b, float* result,
                            size_t n, hipStream_t stream) {
        // For dot product, use cuBLAS (handled in tensor_matrix_ops.cpp)
        // This is just a placeholder
    }

    void launch_uniform(float* data, size_t n, float low, float high,
                        unsigned long long seed, hipStream_t stream) {
        int block_size = 256;
        int grid_size = (n + block_size - 1) / block_size;
        uniform_kernel<<<grid_size, block_size, 0, stream>>>(data, n, low, high, seed);
    }

    void launch_normal(float* data, size_t n, float mean, float std,
                       unsigned long long seed, hipStream_t stream) {
        int block_size = 256;
        int grid_size = (n + block_size - 1) / block_size;
        normal_kernel<<<grid_size, block_size, 0, stream>>>(data, n, mean, std, seed);
    }

    void launch_bernoulli(float* data, size_t n, float p,
                          unsigned long long seed, hipStream_t stream) {
        int block_size = 256;
        int grid_size = (n + block_size - 1) / block_size;
        bernoulli_kernel<<<grid_size, block_size, 0, stream>>>(data, n, p, seed);
    }

    void launch_randint(int* data, size_t n, int low, int high,
                        unsigned long long seed, hipStream_t stream) {
        int block_size = 256;
        int grid_size = (n + block_size - 1) / block_size;
        randint_kernel<<<grid_size, block_size, 0, stream>>>(data, n, low, high, seed);
    }

    void launch_eye(float* data, size_t m, size_t n, hipStream_t stream) {
        size_t total = m * n;
        int block_size = 256;
        int grid_size = (total + block_size - 1) / block_size;
        eye_kernel<<<grid_size, block_size, 0, stream>>>(data, m, n);
    }

    void launch_diag(const float* diagonal, float* matrix, size_t n, hipStream_t stream) {
        size_t total = n * n;
        int block_size = 256;
        int grid_size = (total + block_size - 1) / block_size;
        diag_kernel<<<grid_size, block_size, 0, stream>>>(diagonal, matrix, n);
    }

    void launch_extract_diag(const float* matrix, float* diagonal, size_t n, hipStream_t stream) {
        int block_size = 256;
        int grid_size = (n + block_size - 1) / block_size;
        extract_diag_kernel<<<grid_size, block_size, 0, stream>>>(matrix, diagonal, n);
    }

} // namespace gs::tensor_ops
