#include "hip/hip_runtime.h"
/* SPDX-FileCopyrightText: 2025 LichtFeld Studio Authors
 *
 * SPDX-License-Identifier: GPL-3.0-or-later */

#include "adam.h"
#include "optimizer_config.h"
#include "utils.h"

// Forward declare the kernels (defined in adam_api.cu)
namespace fast_lfs::optimizer::kernels::adam {
    __global__ void adam_step_vectorized_cu(
        float* param,
        float* exp_avg,
        float* exp_avg_sq,
        const float* param_grad,
        const int n_elements,
        const float lr,
        const float beta1,
        const float beta2,
        const float eps,
        const float bias_correction1_rcp,
        const float bias_correction2_sqrt_rcp);
}

void fast_lfs::optimizer::adam_step(
    float* param,
    float* exp_avg,
    float* exp_avg_sq,
    const float* param_grad,
    const int n_elements,
    const float lr,
    const float beta1,
    const float beta2,
    const float eps,
    const float bias_correction1_rcp,
    const float bias_correction2_sqrt_rcp) {

    // Each thread processes 4 elements, so divide by 4
    const int n_threads = div_round_up(n_elements, 4);

    kernels::adam::adam_step_vectorized_cu<<<div_round_up(n_threads, config::block_size_adam_step), config::block_size_adam_step>>>(
        param,
        exp_avg,
        exp_avg_sq,
        param_grad,
        n_elements,
        lr,
        beta1,
        beta2,
        eps,
        bias_correction1_rcp,
        bias_correction2_sqrt_rcp);
    CHECK_CUDA(config::debug, "adam step vectorized")
}
