#include "hip/hip_runtime.h"
/* SPDX-FileCopyrightText: 2025 LichtFeld Studio Authors
 *
 * SPDX-License-Identifier: GPL-3.0-or-later */

#include "lfs/kernels/ssim.cuh"
#include "lfs/kernels/ssim_reduction.cuh"
#include <algorithm>
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>

namespace cg = cooperative_groups;

namespace {
    // ------------------------------------------
    // Constant Memory for Gaussian Coefficients
    // ------------------------------------------
    __constant__ float cGauss[11] = {
        0.001028380123898387f,
        0.0075987582094967365f,
        0.036000773310661316f,
        0.10936068743467331f,
        0.21300552785396576f,
        0.26601171493530273f,
        0.21300552785396576f,
        0.10936068743467331f,
        0.036000773310661316f,
        0.0075987582094967365f,
        0.001028380123898387f};

// ------------------------------------------
// Block and Shared Memory Dimensions
// ------------------------------------------
#define BLOCK_X 16
#define BLOCK_Y 16
#define HALO    5

#define SHARED_X (BLOCK_X + 2 * HALO)
#define SHARED_Y (BLOCK_Y + 2 * HALO)

// For partial results after horizontal pass
#define CONV_X BLOCK_X
#define CONV_Y SHARED_Y

    // ------------------------------------------
    // Utility: Safe pixel fetch w/ zero padding
    // ------------------------------------------
    __device__ __forceinline__ float get_pix_value(
        const float* img,
        int b, int c, int y, int x,
        int CH, int H, int W) {
        if (x < 0 || x >= W || y < 0 || y >= H) {
            return 0.0f;
        }
        return img[b * CH * H * W + c * H * W + y * W + x];
    }

    // ------------------------------------------
    // Forward Kernel: Fused SSIM
    //  - Two-pass convolution to get mu1, mu2,
    //    sigma1_sq, sigma2_sq, sigma12, etc.
    //  - Writes final SSIM map to ssim_map
    //  - Optionally writes partial derivatives
    //    to dm_dmu1, dm_dsigma1_sq, dm_dsigma12
    // ------------------------------------------
    __global__ void fusedssimCUDA(
        int H,
        int W,
        int CH,
        float C1,
        float C2,
        const float* __restrict__ img1,
        const float* __restrict__ img2,
        float* __restrict__ ssim_map,
        float* __restrict__ dm_dmu1,
        float* __restrict__ dm_dsigma1_sq,
        float* __restrict__ dm_dsigma12) {
        auto block = cg::this_thread_block();
        const int bIdx = block.group_index().z; // batch index
        const int pix_y = block.group_index().y * BLOCK_Y + block.thread_index().y;
        const int pix_x = block.group_index().x * BLOCK_X + block.thread_index().x;
        const int pix_id = pix_y * W + pix_x;
        const int num_pix = H * W;

        // Shared memory for the tile (img1, img2)
        __shared__ float sTile[SHARED_Y][SHARED_X][2];
        // After horizontal pass, store partial sums here
        // xconv[y][x] -> (sumX, sumX^2, sumY, sumY^2, sumXY)
        __shared__ float xconv[CONV_Y][CONV_X][5];

        // Each block processes B x C sub-batches. We loop over channels:
        for (int c = 0; c < CH; ++c) {
            // ------------------------------------------------------------
            // 1) Load (img1, img2) tile + halo into shared memory
            // ------------------------------------------------------------
            {
                const int tileSize = SHARED_Y * SHARED_X;
                const int threads = BLOCK_X * BLOCK_Y;
                const int steps = (tileSize + threads - 1) / threads;

                const int tileStartY = block.group_index().y * BLOCK_Y;
                const int tileStartX = block.group_index().x * BLOCK_X;

                for (int s = 0; s < steps; ++s) {
                    int tid = s * threads + block.thread_rank();
                    if (tid < tileSize) {
                        int local_y = tid / SHARED_X;
                        int local_x = tid % SHARED_X;
                        int gy = tileStartY + local_y - HALO;
                        int gx = tileStartX + local_x - HALO;

                        float X = get_pix_value(img1, bIdx, c, gy, gx, CH, H, W);
                        float Y = get_pix_value(img2, bIdx, c, gy, gx, CH, H, W);

                        sTile[local_y][local_x][0] = X;
                        sTile[local_y][local_x][1] = Y;
                    }
                }
            }
            block.sync();

            // ------------------------------------------------------------
            // 2) Horizontal convolution (11x1) in shared memory
            //    We'll accumulate symmetrical pairs around center.
            // ------------------------------------------------------------
            {
                int ly = threadIdx.y;
                int lx = threadIdx.x + HALO; // skip left halo

                float sumX = 0.f;
                float sumX2 = 0.f;
                float sumY = 0.f;
                float sumY2 = 0.f;
                float sumXY = 0.f;

                // #pragma unroll for those 5 pairs
#pragma unroll
                for (int d = 1; d <= HALO; ++d) {
                    float w = cGauss[HALO - d];
                    float Xleft = sTile[ly][lx - d][0];
                    float Yleft = sTile[ly][lx - d][1];
                    float Xright = sTile[ly][lx + d][0];
                    float Yright = sTile[ly][lx + d][1];

                    sumX += (Xleft + Xright) * w;
                    sumX2 += ((Xleft * Xleft) + (Xright * Xright)) * w;
                    sumY += (Yleft + Yright) * w;
                    sumY2 += ((Yleft * Yleft) + (Yright * Yright)) * w;
                    sumXY += ((Xleft * Yleft) + (Xright * Yright)) * w;
                }
                // center
                {
                    float centerX = sTile[ly][lx][0];
                    float centerY = sTile[ly][lx][1];
                    float wc = cGauss[HALO];
                    sumX += centerX * wc;
                    sumX2 += (centerX * centerX) * wc;
                    sumY += centerY * wc;
                    sumY2 += (centerY * centerY) * wc;
                    sumXY += (centerX * centerY) * wc;
                }

                // Write out partial sums
                xconv[ly][threadIdx.x][0] = sumX;
                xconv[ly][threadIdx.x][1] = sumX2;
                xconv[ly][threadIdx.x][2] = sumY;
                xconv[ly][threadIdx.x][3] = sumY2;
                xconv[ly][threadIdx.x][4] = sumXY;

                // Possibly handle second row in same warp
                int ly2 = ly + BLOCK_Y;
                if (ly2 < CONV_Y) {
                    sumX = 0.f;
                    sumX2 = 0.f;
                    sumY = 0.f;
                    sumY2 = 0.f;
                    sumXY = 0.f;

#pragma unroll
                    for (int d = 1; d <= HALO; ++d) {
                        float w = cGauss[HALO - d];
                        float Xleft = sTile[ly2][lx - d][0];
                        float Yleft = sTile[ly2][lx - d][1];
                        float Xright = sTile[ly2][lx + d][0];
                        float Yright = sTile[ly2][lx + d][1];

                        sumX += (Xleft + Xright) * w;
                        sumX2 += ((Xleft * Xleft) + (Xright * Xright)) * w;
                        sumY += (Yleft + Yright) * w;
                        sumY2 += ((Yleft * Yleft) + (Yright * Yright)) * w;
                        sumXY += ((Xleft * Yleft) + (Xright * Yright)) * w;
                    }
                    // center
                    {
                        float cx = sTile[ly2][lx][0];
                        float cy = sTile[ly2][lx][1];
                        float wc = cGauss[HALO];
                        sumX += cx * wc;
                        sumX2 += (cx * cx) * wc;
                        sumY += cy * wc;
                        sumY2 += (cy * cy) * wc;
                        sumXY += (cx * cy) * wc;
                    }
                    xconv[ly2][threadIdx.x][0] = sumX;
                    xconv[ly2][threadIdx.x][1] = sumX2;
                    xconv[ly2][threadIdx.x][2] = sumY;
                    xconv[ly2][threadIdx.x][3] = sumY2;
                    xconv[ly2][threadIdx.x][4] = sumXY;
                }
            }
            block.sync();

            // ------------------------------------------------------------
            // 3) Vertical convolution (1x11) + final SSIM
            // ------------------------------------------------------------
            {
                int ly = threadIdx.y + HALO;
                int lx = threadIdx.x;

                float out0 = 0.f, out1 = 0.f, out2 = 0.f, out3 = 0.f, out4 = 0.f;

#pragma unroll
                for (int d = 1; d <= HALO; ++d) {
                    float w = cGauss[HALO - d];
                    float* top = xconv[ly - d][lx];
                    float* bot = xconv[ly + d][lx];

                    out0 += (top[0] + bot[0]) * w;
                    out1 += (top[1] + bot[1]) * w;
                    out2 += (top[2] + bot[2]) * w;
                    out3 += (top[3] + bot[3]) * w;
                    out4 += (top[4] + bot[4]) * w;
                }
                // center
                {
                    float wC = cGauss[HALO];
                    float* ctr = xconv[ly][lx];
                    out0 += ctr[0] * wC;
                    out1 += ctr[1] * wC;
                    out2 += ctr[2] * wC;
                    out3 += ctr[3] * wC;
                    out4 += ctr[4] * wC;
                }

                if (pix_x < W && pix_y < H) {
                    float mu1 = out0;
                    float mu2 = out2;
                    float mu1_sq = mu1 * mu1;
                    float mu2_sq = mu2 * mu2;

                    float sigma1_sq = out1 - mu1_sq;
                    float sigma2_sq = out3 - mu2_sq;
                    float sigma12 = out4 - mu1 * mu2;

                    float A = mu1_sq + mu2_sq + C1;
                    float B = sigma1_sq + sigma2_sq + C2;
                    float C_ = 2.f * mu1 * mu2 + C1;
                    float D_ = 2.f * sigma12 + C2;

                    float val = (C_ * D_) / (A * B);

                    int global_idx = bIdx * CH * num_pix + c * num_pix + pix_id;
                    ssim_map[global_idx] = val;

                    if (dm_dmu1) {
                        // partial derivatives
                        float d_m_dmu1 = ((mu2 * 2.f * D_) / (A * B) - (mu2 * 2.f * C_) / (A * B) - (mu1 * 2.f * C_ * D_) / (A * A * B) + (mu1 * 2.f * C_ * D_) / (A * B * B));
                        float d_m_dsigma1_sq = (-C_ * D_) / (A * B * B);
                        float d_m_dsigma12 = (2.f * C_) / (A * B);

                        dm_dmu1[global_idx] = d_m_dmu1;
                        dm_dsigma1_sq[global_idx] = d_m_dsigma1_sq;
                        dm_dsigma12[global_idx] = d_m_dsigma12;
                    }
                }
            }
        }
    }

    // ------------------------------------------
    // Backward Kernel: Apply chain rule to get
    //    dL/d(img1) from partial derivatives
    //    (dm_dmu1, dm_dsigma1_sq, dm_dsigma12)
    //    and dL/dmap (the gradient from above).
    // ------------------------------------------
    __global__ void fusedssim_backwardCUDA(
        int H,
        int W,
        int CH,
        float C1,
        float C2,
        const float* __restrict__ img1,
        const float* __restrict__ img2,
        const float* __restrict__ dL_dmap,
        float* __restrict__ dL_dimg1,
        const float* __restrict__ dm_dmu1,
        const float* __restrict__ dm_dsigma1_sq,
        const float* __restrict__ dm_dsigma12) {
        auto block = cg::this_thread_block();

        const int pix_y = block.group_index().y * BLOCK_Y + block.thread_index().y;
        const int pix_x = block.group_index().x * BLOCK_X + block.thread_index().x;
        const int pix_id = pix_y * W + pix_x;
        const int num_pix = H * W;
        const int bIdx = block.group_index().z;

        // Shared memory for the fused data:
        // [0]: dm_dmu1*dL, [1]: dm_dsigma1_sq*dL, [2]: dm_dsigma12*dL
        __shared__ float sData[3][SHARED_Y][SHARED_X];
        __shared__ float sScratch[CONV_Y][CONV_X][3];

        for (int c = 0; c < CH; ++c) {
            float p1 = 0.f, p2 = 0.f;
            if (pix_x < W && pix_y < H) {
                p1 = get_pix_value(img1, bIdx, c, pix_y, pix_x, CH, H, W);
                p2 = get_pix_value(img2, bIdx, c, pix_y, pix_x, CH, H, W);
            }

            // (1) Load + fuse multiplication
            {
                const int start_y = block.group_index().y * BLOCK_Y;
                const int start_x = block.group_index().x * BLOCK_X;

                int tid = threadIdx.y * blockDim.x + threadIdx.x;
                int warp_id = tid / 32;
                int lane_id = tid % 32;
                int totalThreads = BLOCK_X * BLOCK_Y;
                int num_warps = (totalThreads + 31) / 32;

                for (int row = warp_id; row < SHARED_Y; row += num_warps) {
                    int gy = start_y + row - HALO;
                    for (int col = lane_id; col < SHARED_X; col += 32) {
                        int gx = start_x + col - HALO;

                        float chain = get_pix_value(dL_dmap, bIdx, c, gy, gx, CH, H, W);
                        float vmu = get_pix_value(dm_dmu1, bIdx, c, gy, gx, CH, H, W);
                        float vs1 = get_pix_value(dm_dsigma1_sq, bIdx, c, gy, gx, CH, H, W);
                        float vs12 = get_pix_value(dm_dsigma12, bIdx, c, gy, gx, CH, H, W);

                        sData[0][row][col] = vmu * chain;
                        sData[1][row][col] = vs1 * chain;
                        sData[2][row][col] = vs12 * chain;
                    }
                }
            }
            block.sync();

            // (2) Horizontal pass
            {
                int ly = threadIdx.y;
                int lx = threadIdx.x + HALO;

                for (int pass = 0; pass < 2; ++pass) {
                    int yy = ly + pass * BLOCK_Y;
                    if (yy < CONV_Y) {
                        float accum0 = 0.f, accum1 = 0.f, accum2 = 0.f;

#pragma unroll
                        for (int d = 1; d <= HALO; ++d) {
                            float w = cGauss[HALO - d];
                            float left0 = sData[0][yy][lx - d];
                            float left1 = sData[1][yy][lx - d];
                            float left2 = sData[2][yy][lx - d];

                            float right0 = sData[0][yy][lx + d];
                            float right1 = sData[1][yy][lx + d];
                            float right2 = sData[2][yy][lx + d];

                            accum0 += (left0 + right0) * w;
                            accum1 += (left1 + right1) * w;
                            accum2 += (left2 + right2) * w;
                        }
                        // center
                        {
                            float wc = cGauss[HALO];
                            float c0 = sData[0][yy][lx];
                            float c1 = sData[1][yy][lx];
                            float c2 = sData[2][yy][lx];
                            accum0 += c0 * wc;
                            accum1 += c1 * wc;
                            accum2 += c2 * wc;
                        }

                        sScratch[yy][threadIdx.x][0] = accum0;
                        sScratch[yy][threadIdx.x][1] = accum1;
                        sScratch[yy][threadIdx.x][2] = accum2;
                    }
                }
            }
            block.sync();

            // (3) Vertical pass -> finalize dL/d(img1)
            if (pix_x < W && pix_y < H) {
                int ly = threadIdx.y + HALO;
                int lx = threadIdx.x;

                float sum0 = 0.f, sum1 = 0.f, sum2 = 0.f;

#pragma unroll
                for (int d = 1; d <= HALO; ++d) {
                    float w = cGauss[HALO - d];
                    float* top = sScratch[ly - d][lx];
                    float* bot = sScratch[ly + d][lx];

                    sum0 += (top[0] + bot[0]) * w;
                    sum1 += (top[1] + bot[1]) * w;
                    sum2 += (top[2] + bot[2]) * w;
                }
                // center
                {
                    float wc = cGauss[HALO];
                    float* ctr = sScratch[ly][lx];
                    sum0 += ctr[0] * wc;
                    sum1 += ctr[1] * wc;
                    sum2 += ctr[2] * wc;
                }

                // final accumulation
                float dL_dpix = sum0 + (2.f * p1) * sum1 + (p2)*sum2;

                int out_idx = bIdx * CH * num_pix + c * num_pix + pix_id;
                dL_dimg1[out_idx] = dL_dpix;
            }
            block.sync();
        }
    }

} // anonymous namespace

// LibTorch-Free API
namespace lfs::training::kernels {

    std::pair<float, SSIMContext> ssim_forward(
        const lfs::core::Tensor& img1_input,
        const lfs::core::Tensor& img2_input,
        bool apply_valid_padding) {

        const float C1 = 0.01f * 0.01f;
        const float C2 = 0.03f * 0.03f;

        // Make tensors contiguous and ensure 4D [N, C, H, W]
        auto img1 = img1_input.contiguous();
        auto img2 = img2_input.contiguous();

        if (img1.ndim() == 3) {
            img1 = img1.unsqueeze(0);
        }
        if (img2.ndim() == 3) {
            img2 = img2.unsqueeze(0);
        }

        int N = static_cast<int>(img1.shape()[0]);
        int C = static_cast<int>(img1.shape()[1]);
        int H = static_cast<int>(img1.shape()[2]);
        int W = static_cast<int>(img1.shape()[3]);

        // Launch config
        dim3 grid((W + BLOCK_X - 1) / BLOCK_X,
                  (H + BLOCK_Y - 1) / BLOCK_Y,
                  N);
        dim3 block(BLOCK_X, BLOCK_Y);

        // Output SSIM map
        auto ssim_map = lfs::core::Tensor::zeros(img1.shape(), lfs::core::Device::CUDA);

        // Allocate derivative Tensors
        auto dm_dmu1 = lfs::core::Tensor::zeros(img1.shape(), lfs::core::Device::CUDA);
        auto dm_dsigma1_sq = lfs::core::Tensor::zeros(img1.shape(), lfs::core::Device::CUDA);
        auto dm_dsigma12 = lfs::core::Tensor::zeros(img1.shape(), lfs::core::Device::CUDA);

        fusedssimCUDA<<<grid, block>>>(
            H, W, C, C1, C2,
            img1.ptr<float>(),
            img2.ptr<float>(),
            ssim_map.ptr<float>(),
            dm_dmu1.ptr<float>(),
            dm_dsigma1_sq.ptr<float>(),
            dm_dsigma12.ptr<float>());

        // Store original dimensions
        int h = H;
        int w = W;

        // Apply valid padding (crop 5 pixels from each side) using efficient view slicing
        // Then compute mean using optimized tensor reduction (matches PyTorch speed!)
        lfs::core::Tensor ssim_map_cropped = ssim_map;
        if (apply_valid_padding && H > 10 && W > 10) {
            ssim_map_cropped = ssim_map.slice(2, 5, H - 5).slice(3, 5, W - 5);
        }

        // Use tensor library's optimized mean (warp reductions + vectorized loads)
        float ssim_value = ssim_map_cropped.mean().item<float>();

        // Save context for backward
        SSIMContext ctx;
        ctx.img1 = img1;
        ctx.img2 = img2;
        ctx.dm_dmu1 = dm_dmu1;
        ctx.dm_dsigma1_sq = dm_dsigma1_sq;
        ctx.dm_dsigma12 = dm_dsigma12;
        ctx.original_h = h;
        ctx.original_w = w;
        ctx.apply_valid_padding = apply_valid_padding;

        return std::make_pair(ssim_value, std::move(ctx));
    }

    lfs::core::Tensor ssim_backward(
        const SSIMContext& ctx,
        float grad_loss) {

        const float C1 = 0.01f * 0.01f;
        const float C2 = 0.03f * 0.03f;

        // Compute gradient map size (after cropping if applicable)
        int grad_h = ctx.original_h;
        int grad_w = ctx.original_w;
        size_t N = ctx.img1.shape()[0];
        size_t C = ctx.img1.shape()[1];
        size_t numel = N * C * grad_h * grad_w;

        if (ctx.apply_valid_padding && grad_h > 10 && grad_w > 10) {
            grad_h -= 10; // Remove 5 pixels from each side
            grad_w -= 10;
            numel = N * C * grad_h * grad_w;
        }

        // Create gradient map: d(loss)/d(ssim_scalar) = grad_loss
        // d(ssim_scalar)/d(ssim_map[i]) = 1/numel
        // So: d(loss)/d(ssim_map[i]) = grad_loss / numel
        float grad_per_pixel = grad_loss / static_cast<float>(numel);

        // Create gradient tensor for cropped region
        auto dL_dmap = lfs::core::Tensor::zeros(ctx.img1.shape(), lfs::core::Device::CUDA);

        if (ctx.apply_valid_padding && ctx.original_h > 10 && ctx.original_w > 10) {
            // Fill cropped region with gradient
            auto cropped_view = dL_dmap.slice(2, 5, ctx.original_h - 5).slice(3, 5, ctx.original_w - 5);
            cropped_view.fill_(grad_per_pixel);
        } else {
            // No cropping - fill entire map
            dL_dmap.fill_(grad_per_pixel);
        }

        // Allocate output gradient
        auto dL_dimg1 = lfs::core::Tensor::zeros(ctx.img1.shape(), lfs::core::Device::CUDA);

        // Launch backward kernel
        dim3 grid((ctx.original_w + BLOCK_X - 1) / BLOCK_X,
                  (ctx.original_h + BLOCK_Y - 1) / BLOCK_Y,
                  N);
        dim3 block(BLOCK_X, BLOCK_Y);

        fusedssim_backwardCUDA<<<grid, block>>>(
            ctx.original_h, ctx.original_w, static_cast<int>(C), C1, C2,
            ctx.img1.ptr<float>(),
            ctx.img2.ptr<float>(),
            dL_dmap.ptr<float>(),
            dL_dimg1.ptr<float>(),
            ctx.dm_dmu1.ptr<float>(),
            ctx.dm_dsigma1_sq.ptr<float>(),
            ctx.dm_dsigma12.ptr<float>());

        return dL_dimg1;
    }

} // namespace lfs::training::kernels
